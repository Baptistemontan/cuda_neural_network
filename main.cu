#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>
#include <iostream>
#include "util/img.cuh"
#include "neural/nn.cuh"
#include "neural/activations.cuh"


#define SAVE_FILE_NAME "./testing_net/bin"

using fprecision = float;

int main() {

	srand(time(NULL));

	//TRAINING
	printf("training\n");
	size_t number_training_imgs = 10000; // 10_000
	size_t epochs = 40;
	Img* training_imgs;
	if(csv_to_imgs(&training_imgs, "./data/mnist_test.csv", number_training_imgs)) {
		printf("An error happened while loading the imgs.\n");
		exit(EXIT_FAILURE);
	}
	
	std::function<fprecision(const fprecision&)> activation = [](auto x) {
		return relu(x);
	};
	std::function<fprecision(const fprecision&)> activation_prime = [](auto x) {
		return relu_prime(x);
	};

	NeuralNetwork<fprecision, 784, 300, 10> net;

	net.train_batch_cuda<50>(training_imgs, epochs, number_training_imgs, (fprecision)0.5, (fprecision)0.9, ActivationCuda::Relu);

	// std::ofstream output_file(SAVE_FILE_NAME, std::ios::out | std::ios::binary | std::ios::trunc);
	// if(!output_file) {
	// 	throw std::runtime_error("unable to open the output file");
	// }
	// net.save_binary(output_file);
	// output_file.close();
	
	imgs_free(training_imgs, number_training_imgs);

	// PREDICTING
	printf("predicting\n");
	size_t number_test_imgs = 3000;
	Img* test_imgs;
	if(csv_to_imgs(&test_imgs, "data/mnist_test.csv", number_test_imgs)) {
		printf("An error appened while loading the imgs.\n");
		exit(EXIT_FAILURE);
	}

	double score = net.predict_imgs(test_imgs, number_test_imgs, activation);
	printf("Score: %2.3f%%\n", score * 100);
	imgs_free(test_imgs, number_test_imgs);


	return EXIT_SUCCESS;
}